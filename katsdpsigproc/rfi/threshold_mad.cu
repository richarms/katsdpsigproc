#include "hip/hip_runtime.h"
// Note: does not compile as-is: must be run through mako

/**
 * @file
 *
 * Median-of-absolute-deviations thresholder. The current design has some
 * number of threads (in the same workgroup) cooperatively computing the
 * median for each channel. However, the threads in each warp are spread
 * across baselines, to ensure efficient memory accesses.
 *
 * This prevents an entire channel from being loaded into the register
 * file, which makes the implementation bandwidth-heavy. A better
 * approach may be to transpose the data and use a workgroup per
 * channel.
 *
 * Medians are found by a binary search to find the value with the
 * required rank. This is done over binary representation of the
 * floating-point values, exploiting the fact that the IEEE-754 encoding
 * of positive floating-point values has the same ordering as the values
 * themselves.
 */

<%namespace name="rank" file="/rank.cu"/>
<%namespace name="common" file="threshold_mad_common.cu"/>

/**
 * Encapsulates a section of a strided (non-contiguous) 1D array.
 */
typedef struct array_piece
{
    const float *in;
    int start;
    int end;
    int stride;
} array_piece;

__device__ void array_piece_init(
    array_piece *self,
    const float *in, int start, int end, int stride)
{
    self->in = in;
    self->start = start;
    self->end = end;
    self->stride = stride;
}

__device__ float array_piece_get(const array_piece *self, int idx)
{
    return self->in[idx * self->stride];
}

<%rank:ranker_serial class_name="ranker_abs_serial" type="float">
    <%def name="foreach(self)">
        for (int i = (${self})->piece.start; i < (${self})->piece.end; i++)
        {
            ${caller.body('fabs(array_piece_get(&(%s)->piece, i))' % (self,))}
        }
    </%def>
    array_piece piece;
</%rank:ranker_serial>

__device__ void ranker_abs_serial_init(ranker_abs_serial *self, const array_piece *piece)
{
    self->piece = *piece;
}

<%rank:ranker_parallel class_name="ranker_abs_parallel" serial_class="ranker_abs_serial" type="float" size="${wgsy}">
</%rank:ranker_parallel>
__device__ void ranker_abs_parallel_init(
    ranker_abs_parallel *self,
    const array_piece *piece,
    ranker_abs_parallel_scratch *scratch,
    int tid)
{
    ranker_abs_serial_init(&self->serial, piece);
    self->scratch = scratch;
    self->tid = tid;
}

<%common:median_non_zero ranker_class="ranker_abs_parallel"/>

__global__ void __launch_bounds__(${wgsx * wgsy}) threshold_mad(
    const float * __restrict in, unsigned char * __restrict flags,
    int channels, int stride, float factor,
    int VT)
{
    __shared__ ranker_abs_parallel_scratch scratch[${wgsx}];

    int bl = blockDim.x * blockIdx.x + threadIdx.x;
    int start = threadIdx.y * VT;
    int end = min(start + VT, channels);
    array_piece piece;
    array_piece_init(&piece, in + bl, start, end, stride);
    ranker_abs_parallel ranker;
    ranker_abs_parallel_init(&ranker, &piece, scratch + threadIdx.x, threadIdx.y);
    float threshold = factor * median_non_zero(&ranker, channels);
    for (int i = piece.start; i < piece.end; i++)
        flags[bl + i * stride] = (array_piece_get(&piece, i) > threshold) ? ${flag_value} : 0;
}
